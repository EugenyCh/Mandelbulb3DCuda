#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "CImg/CImg.h"
#include "mandMath.cuh"
#include <iostream>
#include <ctime>
#include <iomanip>
#include <string>
#include <sstream>
#include <cmath>

#define PI 3.14159265358979323846

using namespace std;
using namespace cimg_library;

__global__ void kernel(
	unsigned char* buffer,
	int offsetX,
	int offsetY,
	int offsetZ,
	int rectSize,
	int side,
	int iters,
	double power,
	double angleXZ,
	double angleYZ)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	if (x >= rectSize)
		return;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	if (y >= rectSize)
		return;
	x += offsetX;
	if (x >= side)
		return;
	y += offsetY;
	if (y >= side)
		return;
	int offset = x + y * side;
	if (buffer[offset * 3] > 0)
		return;

	// Compute a point at this position
	int side1_2 = side >> 1;
	double bailout = pow(2.0, 1.0 / (power - 1.0));
	double bailout2 = bailout * bailout;
	double cx = (bailout * (x - side1_2)) / side1_2;
	double cy = (bailout * (y - side1_2)) / side1_2;

	// Iterating
	bool belongs = false;
	double sqr;
	int z1 = side - 1 - offsetZ;
	int z2 = max(side1_2 - 1, side - 1 - rectSize - offsetZ);
	for (int z = z1; z >= z2; --z)
	{
		double cz = (bailout * (z - side1_2)) / side1_2;
		cuEuclidVector cVec(cx, cy, cz);
		cVec.rotate(angleXZ, 0, 2);
		cVec.rotate(angleYZ, 1, 2);
		sqr = cVec.square();
		cuEuclidVector vec(cVec);
		for (int i = 0; i < iters; ++i)
			vec = vec.pow(power) + cVec;
		if (vec.square() <= bailout2)
		{
			belongs = true;
			break;
		}
	}

	// Setting the point color
	if (belongs)
	{
		double k = sqr / bailout2;
		buffer[offset * 3] = (unsigned char)(1 + k * 127);
		buffer[offset * 3 + 1] = (unsigned char)(k * 127);
		buffer[offset * 3 + 2] = (unsigned char)((2.0 * k - k * k) * 255);
	}
}

int main(int argc, char** argv)
{
	// Settting
	if (argc != 4 && argc != 6)
	{
		cout << "Usage: MandBulb <side width in pixels> <power> <iterations> [<angleXZ> <angleYZ>]" << endl;
		return 1;
	}

	int side;
	int power;
	int iters;
	double angleXZ = 0.0;
	double angleYZ = 0.0;

	try
	{
		side = stoi(argv[1]);
		if (side < 32)
		{
			cout << "Width must be integer >= 32" << endl;
			return 1;
		}
	}
	catch (...)
	{
		cout << "Width must integer" << endl;
		return 1;
	}

	try
	{
		power = stoi(argv[2]);
		if (power < 2)
		{
			cout << "Power must be integer >= 2" << endl;
			return 1;
		}
	}
	catch (...)
	{
		cout << "Power must be integer" << endl;
		return 1;
	}

	try
	{
		iters = stoi(argv[3]);
		if (iters < 0)
		{
			cout << "Iterations must be integer >= 0" << endl;
			return 1;
		}
	}
	catch (...)
	{
		cout << "Iterations must be integer" << endl;
		return 1;
	}

	if (argc == 6)
	{
		try
		{
			angleXZ = stod(argv[4]);
		}
		catch (...)
		{
			cout << "AngleXZ must be floating-point number" << endl;
			return 1;
		}

		try
		{
			angleYZ = stod(argv[5]);
		}
		catch (...)
		{
			cout << "AngleYZ must be floating-point number" << endl;
			return 1;
		}
	}

	angleXZ = angleXZ * PI / 180.0;
	angleYZ = angleYZ * PI / 180.0;

	// Initializing
	const int sz = side * side;
	unsigned char* buffer = new unsigned char[sz * 3];
	unsigned char* dev_buffer;
	CImg<unsigned char> image(side, side, 1, 3, 0);

	hipError_t status;
	if ((status = hipMalloc((void**)& dev_buffer, sz * 3)) != hipSuccess)
	{
		cerr << "Error on creating buffer of pixels in GPU" << endl;
		return status;
	}

	// Rendering
	time_t tStart = time(0);
	dim3 blocks(5, 5);
	dim3 threads(32, 32);
	int gridDim = (side + 159) / 160;
	int index = 0;
	for (int yi = 0; yi < gridDim; ++yi)
		for (int xi = 0; xi < gridDim; ++xi)
			for (int zi = 0; zi < gridDim; ++zi)
			{
				cout << "\rRendering " << (++index) << " / " << (gridDim * gridDim * gridDim);
				kernel<<<blocks, threads>>> (
					dev_buffer,
					xi * 160,
					yi * 160,
					zi * 160,
					160,
					side,
					iters,
					power,
					angleXZ,
					angleYZ);
				hipDeviceSynchronize();
			}

	time_t tFinish = time(0);
	double tDelta = difftime(tFinish, tStart);
	cout << "\nIt tooks " << setprecision(3) << showpoint << tDelta << " seconds" << endl;

	// Copying device buffer to host
	cout << "Moving" << endl;
	if ((status = hipMemcpy((void*)buffer, dev_buffer, sz * 3, hipMemcpyDeviceToHost)) != hipSuccess)
	{
		cerr << "Error on getting buffer of pixels from GPU: " << status << endl;
		return status;
	}

	// Freeing
	hipFree(dev_buffer);

	// Filling
	cout << "Filling" << endl;
	for (int y = 0; y < side; ++y)
		for (int x = 0; x < side; ++x)
			image.draw_point(x, y, &buffer[3 * (y * side + x)]);

	// Saving
	stringstream filename;
	filename << "Mandelbulb-"
		<< side << "x" << side
		<< "-n" << power
		<< "-i" << iters
		<< "-xz" << (int)(angleXZ * 180.0 / PI)
		<< "-yz" << (int)(angleYZ * 180.0 / PI)
		<< ".bmp";
	cout << "Saving to " << filename.str() << endl;
	image.save_bmp(filename.str().c_str());

	return 0;
}
